#include "hip/hip_runtime.h"
#include "cudaMetropolizer.hpp"
#include "discretizer.hpp"

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

template <int dim, class su2Type>
__global__ void kernel_probeSite(su2Action<dim> act, su2Type *fields,
                                 CUDA_RAND_STATE_TYPE *randStates,
                                 int *hitCounts, int multiProbe, double delta,
                                 int odd, int mu) {

  int idx = (threadIdx.x + blockDim.x * blockIdx.x);
  int site = 2 * idx;
  int offset = 0;
  for (int i = 0; i < dim; i++) {
    offset += site / act.getBasis(i);
  }

  site += ((offset + odd) % 2);

  if (site >= act.getSiteCount()) {
    return;
  }

  int loc = (dim * site) + mu;
  for (int i = 0; i < multiProbe; i++) {
    su2Type newElement = fields[loc].randomize(delta, &randStates[idx]);
    double change =
        act.template evaluateDelta<su2Type>(fields, newElement, site, mu);
    if ((change < 0) ||
        (hiprand_uniform_double(&randStates[idx]) < exp(-change))) {
      fields[loc] = newElement;
      hitCounts[idx]++;
    }
  }
  fields[loc].renormalize();
}

template <int dim, class su2Type>
__global__ void kernel_measurePlaquette(double *sumBuffer, int *hitBuffer,
                                        su2Type *fields, int *hitCounts,
                                        su2Action<dim> action,
                                        int sitesPerThread) {
  const int tid = threadIdx.x;

  sumBuffer[tid] = 0;
  hitBuffer[tid] = 0;

  for (int i = 0; i < sitesPerThread; i++) {
    int site = (sitesPerThread * tid) + i;
    if (site < action.getSiteCount()) {
      for (int mu = 0; mu < dim; mu++) {
        for (int nu = 0; nu < mu; nu++) {
          sumBuffer[tid] +=
              action.plaquetteProduct(fields, site, mu, nu).trace();
        }
      }
      if (site % 2 == 0) {
        hitBuffer[tid] += hitCounts[site / 2];
        hitCounts[site / 2] = 0;
      }
    }
  }

  int stepSize = 1;
  int activeThreads = CUDA_BLOCK_SIZE / 2;

  while (activeThreads > 0) {
    __syncthreads();
    if (tid < activeThreads) {
      int fst = tid * stepSize * 2;
      int snd = fst + stepSize;
      sumBuffer[fst] += sumBuffer[snd];
      hitBuffer[fst] += hitBuffer[snd];
    }

    stepSize *= 2;
    activeThreads /= 2;
  }
}

template <class su2Type>
__global__ void kernel_initIteration(CUDA_RAND_STATE_TYPE *states,
                                     int *hitCounts, int nMax) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ((2 * idx) < nMax) {
    hitCounts[idx] = 0;
    hiprand_init(42, idx, 0, &states[idx]);
  }
}

template <class su2Type>
__global__ void kernel_initFieldsHot(CUDA_RAND_STATE_TYPE *states,
                                     su2Type *fields, int dim, int nMax) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ((2 * idx) < nMax) {
    for (int mu = 0; mu < dim; mu++) {
      for (int i = 0; i < 2; i++) {
        int loc = (dim * ((2 * idx) + i)) + mu;
        fields[loc] = fields[loc].randomize(1.0, &states[idx]);
      }
    }
  }
}

template <int dim, class su2Type>
cudaMetropolizer<dim, su2Type>::cudaMetropolizer(su2Action<dim> iAction,
                                                 int iMultiProbe, double iDelta,
                                                 su2Type *iFields)
    : action(iAction) {
  delta = iDelta;
  multiProbe = iMultiProbe;

  blockCount =
      ((action.getSiteCount() / 2) + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;


  hipMalloc(&randStates,
             sizeof(CUDA_RAND_STATE_TYPE) * (action.getSiteCount() / 2));
  hipMalloc(&hitCounts, sizeof(int) * (action.getSiteCount() / 2));
  fields = iFields;

  kernel_initIteration<su2Type><<<blockCount, CUDA_BLOCK_SIZE>>>(
      randStates, hitCounts, action.getSiteCount());
}

template <int dim, class su2Type>
cudaMetropolizer<dim, su2Type>::~cudaMetropolizer() {
  hipFree(randStates);
  hipFree(hitCounts);
}

template <int dim, class su2Type>
double cudaMetropolizer<dim, su2Type>::sweep(int sweeps) {
  for (int i = 0; i < sweeps; i++) {
    for (int odd = 0; odd < 2; odd++) {
      for (int mu = 0; mu < dim; mu++) {
        kernel_probeSite<dim, su2Type><<<blockCount, CUDA_BLOCK_SIZE>>>(
            action, fields, randStates, hitCounts, multiProbe, delta, odd, mu);
      }
    }
  }
  return this->measurePlaquette();
}

template <int dim, class su2Type>
double cudaMetropolizer<dim, su2Type>::measurePlaquette() {
  int sitesPerThread =
      (action.getSiteCount() + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
  double *sumBuffer;
  int *hitBuffer;

  hipMalloc(&sumBuffer, sizeof(double) * CUDA_BLOCK_SIZE);
  hipMalloc(&hitBuffer, sizeof(int) * CUDA_BLOCK_SIZE);

  kernel_measurePlaquette<dim, su2Type><<<1, CUDA_BLOCK_SIZE>>>(
      sumBuffer, hitBuffer, fields, hitCounts, action, sitesPerThread);

  double out;
  int hitCount;

  hipMemcpy(&out, sumBuffer, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&hitCount, hitBuffer, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(sumBuffer);
  hipFree(hitBuffer);

  hitRate = (double)hitCount / (action.getSiteCount() * dim * multiProbe);
  out /= action.getSiteCount() * dim * (dim - 1);

  return out;
}

template <int dim, class su2Type>
double cudaMetropolizer<dim, su2Type>::getHitRate() {
  return hitRate;
}
