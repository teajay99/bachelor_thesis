#include "hip/hip_runtime.h"
#include "cudaMetropolizer.hpp"
#include <hip/hip_runtime_api.h>

#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

#define CUDA_CALL(x)                                                           \
  do {                                                                         \
    if ((x) != hipSuccess) {                                                  \
      printf("Error at %s:%d\n", __FILE__, __LINE__);                          \
    }                                                                          \
  } while (0)

#define CURAND_CALL(x)                                                         \
  do {                                                                         \
    if ((x) != HIPRAND_STATUS_SUCCESS) {                                        \
      printf("Error at %s:%d\n", __FILE__, __LINE__);                          \
    }                                                                          \
  } while (0)

void checkCudaErrors(int i) {
  hipError_t err = hipGetLastError(); // add
  if (err != hipSuccess) {
    std::cout << "CUDA error " << i << ": " << hipGetErrorString(err)
              << std::endl; // add
    hipProfilerStop();
    exit(1);
  }
}

template <int dim>
__global__ void kernel_probe_site(su2Action<dim> act, su2Element *fields,
                                  CUDA_RAND_STATE_TYPE *randStates,
                                  int *hitCounts, int multiProbe, double delta,
                                  int odd, int mu) {

  int idx = (threadIdx.x + blockDim.x * blockIdx.x);
  int site = 2 * idx;
  int offset = 0;
  for (int i = 0; i < act.getDim(); i++) {
    offset += site / act.getBasis(i);
  }
  site += (offset + odd) % 2;

  if (site >= act.getSiteCount()) {
    return;
  }

  int loc = (dim * site) + mu;
  for (int i = 0; i < multiProbe; i++) {

    // Evaluates action "around" link Variable U_mu (site)
    double oldVal = act.evaluateDelta(fields, site, mu);
    su2Element oldElement = fields[loc];
    fields[loc] = oldElement.randomize(delta, &randStates[idx]);

    // Evaluating action with new link Variable
    double newVal = act.evaluateDelta(fields, site, mu);

    // Deciding wether to keep the new link Variable
    if ((newVal > oldVal) &&
        (hiprand_uniform_double(&randStates[idx]) > exp(-(newVal - oldVal)))) {
      fields[loc] = oldElement;
    } else {
      hitCounts[idx]++;
    }
  }
  fields[loc].renormalize();
}

template <int dim>
__global__ void kernel_measurePlaquette(double *sumBuffer, int *hitBuffer,
                                        su2Element *fields, int *hitCounts,
                                        su2Action<dim> action,
                                        int sitesPerThread) {
  const int tid = threadIdx.x;

  sumBuffer[tid] = 0;
  hitBuffer[tid] = 0;

  for (int i = 0; i < sitesPerThread; i++) {
    int site = (sitesPerThread * tid) + i;
    if (site < action.getSiteCount()) {
      for (int mu = 0; mu < dim; mu++) {
        for (int nu = 0; nu < mu; nu++) {
          sumBuffer[tid] +=
              action.plaquetteProduct(fields, site, mu, nu).trace();
        }
      }
      if (site % 2 == 0) {
        hitBuffer[tid] += hitCounts[site / 2];
        hitCounts[site / 2] = 0;
      }
    }
  }

  int stepSize = 1;
  int activeThreads = CUDA_BLOCK_SIZE / 2;

  while (activeThreads > 0) {
    __syncthreads();
    if (tid < activeThreads) {
      int fst = tid * stepSize * 2;
      int snd = fst + stepSize;
      sumBuffer[fst] += sumBuffer[snd];
      hitBuffer[fst] += hitBuffer[snd];
    }

    stepSize *= 2;
    activeThreads /= 2;
  }
}

__global__ void kernel_initIteration(CUDA_RAND_STATE_TYPE *states,
                                     su2Element *fields, int *hitCounts,
                                     int nMax, int dim) {

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ((2 * idx) < nMax) {
    hitCounts[idx] = 0;
    hiprand_init(42, idx, 0, &states[idx]);
    for (int i = 0; i < 2; i++) {
      for (int mu = 0; mu < dim; mu++) {
        int loc = (dim * ((2 * idx) + i)) + mu;
        fields[loc] = su2Element();
      }
    }
  }
}

__global__ void kernel_initFieldsHot(CUDA_RAND_STATE_TYPE *states,
                                     su2Element *fields, int dim, int nMax) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ((2 * idx) < nMax) {
    for (int mu = 0; mu < dim; mu++) {
      for (int i = 0; i < 2; i++) {
        int loc = (dim * ((2 * idx) + i)) + mu;
        fields[loc] = fields[loc].randomize(1.0, &states[idx]);
      }
    }
  }
}

template <int dim>
cudaMetropolizer<dim>::cudaMetropolizer(su2Action<dim> iAction, int iMultiProbe,
                                        double iDelta, bool cold)
    : action(iAction) {
  delta = iDelta;
  blockCount =
      ((action.getSiteCount() / 2) + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
  multiProbe = iMultiProbe;

  randStateCount = (blockCount + 200 - 1) / 200;

  hipMalloc(&randStates,
             sizeof(CUDA_RAND_STATE_TYPE) * (action.getSiteCount() / 2));
  hipMalloc(&hitCounts, sizeof(int) * (action.getSiteCount() / 2));
  hipMalloc(&fields, sizeof(su2Element) * action.getSiteCount() * dim);

  kernel_initIteration<<<blockCount, CUDA_BLOCK_SIZE>>>(
      randStates, fields, hitCounts, action.getSiteCount(), dim);

  kernel_initFieldsHot<<<blockCount, CUDA_BLOCK_SIZE>>>(randStates, fields, dim,
                                                        action.getSiteCount());
  checkCudaErrors(2);
}

template <int dim> cudaMetropolizer<dim>::~cudaMetropolizer() {
  hipFree(randStates);
  hipFree(fields);
  hipFree(hitCounts);
}

template <int dim> double cudaMetropolizer<dim>::sweep() {
  for (int odd = 0; odd < 2; odd++) {
    for (int mu = 0; mu < dim; mu++) {
      checkCudaErrors(3);
      kernel_probe_site<<<blockCount, CUDA_BLOCK_SIZE>>>(
          action, fields, randStates, hitCounts, multiProbe, delta, odd, mu);
      checkCudaErrors(1);
    }
  }
  int sitesPerThread =
      (action.getSiteCount() + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
  double *sumBuffer;
  int *hitBuffer;

  hipMallocManaged(&sumBuffer, sizeof(double) * CUDA_BLOCK_SIZE);
  hipMallocManaged(&hitBuffer, sizeof(int) * CUDA_BLOCK_SIZE);

  kernel_measurePlaquette<dim><<<1, CUDA_BLOCK_SIZE>>>(
      sumBuffer, hitBuffer, fields, hitCounts, action, sitesPerThread);

  double out;
  int hitCount;

  hipMemcpy(&out, sumBuffer, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&hitCount, hitBuffer, sizeof(int), hipMemcpyDeviceToHost);

  hipFree(sumBuffer);
  hipFree(hitBuffer);

  hitRate = (double)hitCount / (action.getSiteCount() * dim * multiProbe);
  out /= action.getSiteCount() * dim * (dim - 1);

  return out;
}

template <int dim> double cudaMetropolizer<dim>::getHitRate() {
  return hitRate;
}
